#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

typedef struct 
{
	int width;
	int height;
	float* elements;
} Matrix;

#define BLOCK_SIZE 16

__global__ void matmul_kernel(Matrix A, Matrix B, Matrix C);

// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**)&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**)&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	// Allocation C in device memory
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc((void**)&d_C.elements, size);

	// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	matmul_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main(int argc, char *argv[])
{
	Matrix A, B, C;
	A.width = 128; A.height = 64;
	B.width = 64; B.height = 256;
	C.width = B.width; C.height = A.height;
	FILE *fp = fopen("result.txt", "w");
	if (fp == NULL)
	{
		printf("Can't open file!\n");
		exit(EXIT_FAILURE);
	}
	// Allocate memory for Matrix
	size_t size = A.width * A.height * sizeof(float);
	A.elements = (float*)malloc(size);
	size = B.width * B.height * sizeof(float);
	B.elements = (float*)malloc(size);
	size = C.width * C.height * sizeof(float);
	C.elements = (float*)malloc(size);

	// Fill data for A and B
	for (int r = 0; r < A.height; ++r)
	{
		for (int c = 0; c < A.width; ++c)
		{
			A.elements[r * A.width + c] = 3.0;
		}
	}
	for (int r = 0; r < B.height; ++r)
	{
		for (int c = 0; c < B.width; ++c)
		{
			B.elements[r * B.width + c] = 4.0;
		}
	}

	MatMul(A, B, C);

	for (int r = 0; r < C.height; ++r)
	{
		for (int c = 0; c < C.width; ++c)
			fprintf(fp, "%3.1lf\t", C.elements[r * C.width + c]);
		fprintf(fp, "\n");
	}	

	fclose(fp);
	free(A.elements);
	free(B.elements);
	free(C.elements);
}

__global__ void matmul_kernel(Matrix A, Matrix B, Matrix C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[e + row * A.width] 
				  * B.elements[col + e * B.width]; 

	C.elements[row * C.width + col] = Cvalue;
}

