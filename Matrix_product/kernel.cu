#include "hip/hip_runtime.h"
#include <stdio.h>

typedef struct 
{
	int width;
	int height;
	int stride;
	float* elements;
} Matrix;

__global__ void matmul_kernel(Matrix a, Matrix b, Matrix res);

int main()
{
	float *m1, *m2;	// Matrix 1 & 2
	float *dev_m1, *dev_m2;
	float *resultMatrix, dev_resultMatrix;


}

__global__ void matmul_kernel(Matrix a, Matrix b, Matrix res)
{

}

